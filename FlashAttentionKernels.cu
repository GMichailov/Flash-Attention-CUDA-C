#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <hip/hip_runtime_api.h>
#include <cmath>
#include <cuda/pipeline>

#define WARP 32
#define FLOAT_SIZE 4

template<int TILE_SIZE>
__global__ void asyncBufferLoad(const float* __restrict__ matrix, float* __restrict__ matrixSmem, int offset) {
    extern __shared__ float smem[];
    auto pipe = cuda::make_pipeline();
    int thread = threadIdx.x;
    const float4* global_mem = reinterpret_cast<const float4*>(matrix + offset + thread);
    
}
    
template<int DHEAD, int BLOCK_Q_ROWS, int ROWS_PER_WARP>
__global__ void causalFlashAttention(
    const float* __restrict__ Q, const float* __restrict__ K, const float* __restrict__ V,
    const float* __restrict__ L, const float* __restrict__ M,
    int batchSize, int numHeads,
    int seqLenQ, int seqLenK,
    int strideBatchQ, int strideBatchK, int strideBatchV, int strideBatchO,
    int strideHeadQ, int strideHeadK, int strideHeadV, int strideHeadO,
    float scale, 
    int BLOCK_KV_ROWS
)
{
    // Load Q tile
    int warpId = threadIdx.x / WARP;
    int thread = threadIdx.x % 32;
    const float* tile_start = Q + blockIdx.z * strideBatchQ + blockIdx.y * strideHeadQ + blockIdx.x * BLOCK_Q_ROWS;
    constexpr int fragmentSize = DHEAD * BLOCK_Q_ROWS * ROWS_PER_WARP / WARP;
    const float* fragmentStart = tile_start + warpId * ROWS_PER_WARP * DHEAD + thread * fragmentSize;
    float QFrag[fragmentSize];

    // Do reads of float4 and only save what's necessary
    // Logic: Have one thread responsible for float4 and if smaller, calculate how many threads should idle meanwhile.
    if (fragmentSize >= 4 || thread % 4 == 0) {
        #pragma unroll
        for (int reads = 0; reads < fragmentSize; reads+=4) {
            float4 frag = *((const float4*)(fragmentStart + reads));
            int writes = std::min(fragmentSize - reads, 4);
            if (writes > 0) QFrag[reads + 0] = frag.x;
            if (writes > 1) QFrag[reads + 1] = frag.y;
            if (writes > 2) QFrag[reads + 2] = frag.z;
            if (writes > 3) QFrag[reads + 3] = frag.w;
        }
    }
    __syncthreads();    
}