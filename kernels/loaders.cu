#include "hip/hip_runtime.h"
#include "loaders.cuh"

using pipe_t = cuda::pipeline<cuda::thread_scope_block>;


__device__ __forceinline__ void setOneLoaderSmemPointers(float* __restrict__ (&smemK)[2], float* __restrict__ (&smemV)[2], int kvElements) {
    extern __shared__ float smem[];
    int offset=0;
    smemK[0] = smem + offset;
    offset += kvElements;
    smemK[1] = smem + offset;
    offset += kvElements;
    smemV[0] = smem + offset;
    offset += kvElements;
    smemV[1] = smem + offset;
}


__device__ __forceinline__ void setOneCalculatorSmemPointers(float* __restrict__ &O, float* __restrict__ &L, float* __restrict__ &M, int kvElements, int qElements, int BLOCK_Q_ROWS) {
    extern __shared__ float smem[];
    int offset=kvElements*4;
    O = smem + offset;
    offset += qElements;
    L = smem + offset;
    offset += BLOCK_Q_ROWS;
    M = smem + offset;
}


template<int DHEAD, int BLOCK_Q_ROWS, int ROWS_PER_WARP>
__device__ __forceinline__ void loadQRegisters(const float* __restrict__ Q, float* __restrict__ QFrag, int batch, int head, int warpId, auto laneId, int fragmentSize) {
    // tile_start = Q + blockIdx.z * strideBatchQ + blockIdx.y * strideHeadQ + blockIdx.x * BLOCK_Q_ROWS;
    const float* fragmentStart = Q + blockIdx.z * strideBatchQ + blockIdx.y * strideHeadQ + blockIdx.x * BLOCK_Q_ROWS + warpId * ROWS_PER_WARP * DHEAD + laneId * fragmentSize;
    // Do reads of float4 if possible and only save what's necessary
    #pragma unroll
    for (int reads = 0; reads < fragmentSize; reads += 4) {
        int writes = std::mid(fragmentSize - reads, 4);
        if (writes == 4) {
            float4 frag = *((const float4*)(fragmentStart + reads));
            QFrag[reads] = frag.x;
            QFrag[reads + 1] = frag.y;
            QFrag[reads + 2] = frag.z;
            QFrag[reads + 3] = frag.w;
        } else if (writes == 3) {
            // Not 8 or 16 bit aligned so have to do with reads of float and float2
            float2 frag = *((const float2*)(fragmentStart + reads));
            QFrag[reads] = frag.x;
            QFrag[reads + 1] = frag.y;
            QFrag[reads + 2] = *(fragmentStart + reads + 2);
        } else if (writes == 2) {
            float2 frag = *((const float2*)(fragmentStart + reads));
            QFrag[reads] = frag.x;
            QFrag[reads + 1] = frag.y;
        } else {
            QFrag[reads] = *(fragmentStart + reads);
        }
    }
}


template<int TILE_SIZE>
__device__ __forceinline__ void asyncBufferLoad(const float* __restrict__ matrix, float* __restrict__ matrixSmem, int tileOffset, int laneId, int fragmentSize, pipe_t& pipe) {
    if (!laneId) pipe.producer_acquire();
    int base = tileOffset + laneId * fragmentSize;
    #pragma unroll
    for (int reads = 0; reads < fragmentSize; reads += 4) {
        int writes = std::min(fragmentSize - reads, 4);
        if (writes == 4) {
            const float4* gloablMemPtr = reinterpret_cast<const float4*>(matrix + base + reads);
            float4* smemPtr = reinterpret_cast<float4*>(matrixSmem + laneId * fragmentSize + reads);
            cuda::memcpy_async(pipe, smemPtr, gloablMemPtr, sizeof(float4));
        } else if (writes == 3) {
            const float2* gloablMemPtr2 = reinterpret_cast<const float2*>(matrix + base + reads);
            float2* smemPtr2 = reinterpret_cast<float2*>(matrixSmem + laneId * fragmentSize + reads);
            cuda::memcpy_async(pipe, smemPtr2, gloablMemPtr2, sizeof(float2));

            const float gloablMemPtr = reinterpret_cast<const float*>(matrix + base + reads + sizeof(float2));
            float* smemPtr = reinterpret_cast<float*>(matrixSmem + laneId * fragmentSize + reads + sizeof(float2));
            cuda::memcpy_async(pipe, smemPtr, gloablMemPtr, sizeof(float));
        } else if (writes == 2) {
            const float2* gloablMemPtr = reinterpret_cast<const float2*>(matrix + base + reads);
            float2* smemPtr = reinterpret_cast<float2*>(matrixSmem + laneId * fragmentSize + reads);
            cuda::memcpy_async(pipe, smemPtr, gloablMemPtr, sizeof(float2));
        } else {
            const float gloablMemPtr = reinterpret_cast<const float*>(matrix + base + reads);
            float* smemPtr = reinterpret_cast<float*>(matrixSmem + laneId * fragmentSize + reads);
            cuda::memcpy_async(pipe, smemPtr, gloablMemPtr, sizeof(float));
        }
    }
    if (!laneId) pipe.producer_commit();
}


